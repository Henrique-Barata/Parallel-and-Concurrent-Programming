#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define GRAPH_SIZE 2000

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}

__global__ void run_gpu(int graph_size, int k, int *output) {
 
  int i = blockIdx.x* blockDim.x + threadIdx.x;
  int j = blockIdx.y* blockDim.y + threadIdx.y;

  if (i < graph_size && j < graph_size) {
      if (D(i, k) + D(k, j) < D(i, j)) {
        D(i, j) = D(i, k) + D(k, j);
      }                    
  }

}

void floyd_warshall_gpu(const int *graph, int graph_size, int *output) {

  int *device_output;

  hipMalloc(&device_output, sizeof(int) * graph_size * graph_size);

  hipMemcpy(device_output, graph, sizeof(int) * graph_size * graph_size, hipMemcpyHostToDevice);

  dim3 dim_Block (250,250);
  dim3 dim_Thread (8,8);

  for (int k = 0; k < GRAPH_SIZE; k++) {
    run_gpu<<<dim_Block, dim_Thread>>>(graph_size,k,device_output);
  }

  hipMemcpy(output,device_output,sizeof(int) * graph_size * graph_size,hipMemcpyDeviceToHost);

  hipFree(device_output);
}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

int main(int argc, char **argv) {
#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  struct timeval tv1, tv2, tv;
  float time_delta;

  int *graph, *output_cpu, *output_gpu;
  int size;

  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  fprintf(stderr, "running on cpu...\n");
  TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  TIMER_START();
  floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  if (memcmp(output_cpu, output_gpu, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  }

  return 0;
}
